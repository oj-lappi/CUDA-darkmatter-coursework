#include "hip/hip_runtime.h"
/*
A program to calculate a statistic for galaxy distributions
*/
#include <stdint.h>
#include <stdio.h>
#include "file_io.h"
#include "types.h"
#include "angles.h"
#include "cuda_macros.h"
#include <hip/hip_runtime_api.h>
#include <nvToolsExt.h>

#define SET_DIM  100000
#define SAME_DIM 4999950000

#define MIN(X, Y) (((X) < (Y)) ? (X) : (Y))

#define BIN_DIM 360

#define ANGLE_PER_THREAD 5

__global__
void calc_DR_bins(
	cart_coordinate *D,
	cart_coordinate *R,
	BIN_TYPE *bins
)
{
	uint64_t t_id_in_block = threadIdx.y*blockDim.x+threadIdx.x;
	uint64_t x_angle_start = blockDim.x*blockIdx.x*ANGLE_PER_THREAD;
	uint64_t y_angle_start = blockDim.y*blockIdx.y*ANGLE_PER_THREAD;
	
	__shared__ uint32_t block_bins[BIN_DIM];
	__shared__ cart_coordinate shD[ANGLE_PER_THREAD*32];
	__shared__ cart_coordinate shR[ANGLE_PER_THREAD*32];
	if (threadIdx.y == 31){
		for (int i = threadIdx.x;i < 32*ANGLE_PER_THREAD && x_angle_start +i < 100000;i+=32){
			shD[i] = D[x_angle_start+i];
		}
	}else if (threadIdx.y == 30){
		for (int i = threadIdx.x;i < 32*ANGLE_PER_THREAD && y_angle_start +i < 100000;i+=32){
			shR[i] = R[y_angle_start+i];
		}
	}else if (t_id_in_block < BIN_DIM){
        	block_bins[t_id_in_block] = 0;
	}
	__syncthreads();
	for(int i = threadIdx.x;i <  32*ANGLE_PER_THREAD && x_angle_start +i < 100000;i+=32){
		for(int j = threadIdx.y;j < 32*ANGLE_PER_THREAD && y_angle_start + j < 100000;j+=32){
			uint16_t bin_id = (uint16_t)(acosf(
				shD[i].x*shR[j].x+
				shD[i].y*shR[j].y+
				shD[i].z*shR[j].z
			)*RAD2QUARTDEG_CONV_RATE);	
			bin_id = MIN(bin_id,359);
			atomicAdd_block(&block_bins[bin_id],1);
		}
	}

	__syncthreads();

	if (t_id_in_block < BIN_DIM){
		atomicAdd(&bins[t_id_in_block],block_bins[t_id_in_block]);
        }
}

int main(int argc, char *argv[]) {
	if (argc < 3){printf("Too few arguments, need 2\n");return 1;}
	hipFree(0);
	nvtxRangePush(__FUNCTION__);
	nvtxMark("Allocate");
	cart_coordinate *real_coords;
	BIN_TYPE *h_dr_bins;
	BIN_TYPE *h_dd_bins;
	BIN_TYPE *h_rr_bins;
	
	real_coords = (cart_coordinate*)calloc(100000*2,sizeof(cart_coordinate));
	h_dr_bins = (BIN_TYPE *)calloc(BIN_DIM,sizeof(BIN_TYPE));
	h_rr_bins = (BIN_TYPE *)calloc(BIN_DIM,sizeof(BIN_TYPE));
	h_dd_bins = (BIN_TYPE *)calloc(BIN_DIM,sizeof(BIN_TYPE));

	cart_coordinate *d_real_coords;
	BIN_TYPE *d_dr_bins;
	BIN_TYPE *d_dd_bins;
	BIN_TYPE *d_rr_bins;
	nvtxRangePop();

	hipMalloc(&d_real_coords,SET_DIM*2*sizeof(cart_coordinate));
	hipMalloc(&d_dr_bins,BIN_DIM*sizeof(BIN_TYPE));
	hipMalloc(&d_dd_bins,BIN_DIM*sizeof(BIN_TYPE));
	hipMalloc(&d_rr_bins,BIN_DIM*sizeof(BIN_TYPE));
	
	cart_coordinate *d_fake_coords = d_real_coords+SET_DIM;
	unsigned int grid_len =((3125)+ANGLE_PER_THREAD-1)/ANGLE_PER_THREAD;
	dim3 grid_dims(grid_len,grid_len);
	dim3 block_dims(32,32);

	hipStream_t stream[2];
	for (int i = 0; i < 2; ++i)
		hipStreamCreate(&stream[i]);
		
	//Read one file, start DD
	nvtxRangePush(__FUNCTION__);
	nvtxMark("Read real coords");

	galactic_coordinate *raw_coords = (galactic_coordinate *)calloc(SET_DIM,sizeof(galactic_coordinate));
	read_dataset(raw_coords,argv[1],SET_DIM);
	preprocess_cart(raw_coords,real_coords,SET_DIM);
	nvtxRangePop();

	hipMemcpyAsync(d_real_coords,real_coords,SET_DIM*sizeof(cart_coordinate),hipMemcpyHostToDevice,stream[0]);
	gpuErrchk(hipPeekAtLastError());
 	calc_DR_bins<<<grid_dims, block_dims,0,stream[0]>>>(d_real_coords,d_real_coords,d_dd_bins);
	gpuErrchk(hipPeekAtLastError());
	
	//While the first kernel runs, read second file, start RR
	nvtxRangePush(__FUNCTION__);
	nvtxMark("Read random coords");

	read_dataset(raw_coords,argv[2],SET_DIM);
	preprocess_cart(raw_coords,real_coords+SET_DIM,SET_DIM);

	hipMemcpyAsync(d_fake_coords,real_coords+SET_DIM,SET_DIM*sizeof(cart_coordinate),hipMemcpyHostToDevice,stream[1]);
	gpuErrchk(hipPeekAtLastError());
	
	nvtxRangePop();
	
	hipStreamSynchronize(stream[1]);
 	calc_DR_bins<<<grid_dims, block_dims,0,stream[1]>>>(d_fake_coords,d_fake_coords,d_rr_bins);
	gpuErrchk(hipPeekAtLastError());
	
	//Finally, run DR
 	calc_DR_bins<<<grid_dims, block_dims,0,stream[0]>>>(d_real_coords,d_fake_coords,d_dr_bins);
	gpuErrchk(hipPeekAtLastError());

	hipMemcpyAsync(h_dd_bins,d_dd_bins,BIN_DIM*sizeof(BIN_TYPE),hipMemcpyDeviceToHost);
	hipMemcpyAsync(h_rr_bins,d_rr_bins,BIN_DIM*sizeof(BIN_TYPE),hipMemcpyDeviceToHost);
	hipMemcpyAsync(h_dr_bins,d_dr_bins,BIN_DIM*sizeof(BIN_TYPE),hipMemcpyDeviceToHost);
	
        for (int i = 0; i < 2; ++i)
                hipStreamDestroy(stream[i]);

	double kolm_smir[BIN_DIM];
	nvtxRangePush(__FUNCTION__);
	nvtxMark("Calculate statistic");
	calc_kolmogorov(kolm_smir, h_dr_bins, h_dd_bins, h_rr_bins);
	nvtxRangePop();
	nvtxRangePush(__FUNCTION__);
	nvtxMark("Write statistic");
	write_kolmogorov("out/distribution.tsv",kolm_smir,BIN_DIM);
	nvtxRangePop();

	check_sum(h_dr_bins,10000000000);
	check_sum(h_rr_bins,10000000000);
	check_sum(h_dd_bins,10000000000);
	check_values(h_dr_bins);
	check_kolmogorov(kolm_smir);

	hipFree(d_dr_bins);
	hipFree(d_dd_bins);
	hipFree(d_rr_bins);
	hipFree(d_real_coords);
}

